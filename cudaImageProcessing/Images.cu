#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <Math.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <string>
#include <cstring>
__global__ void GBlur(const unsigned char* const Input, unsigned char* const Output, int numRows, int numCols, const float* const filter, const int Width)
{
    int cols = blockIdx.x * blockDim.x + threadIdx.x;
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    if (cols >= numCols || rows >= numRows)
    {
        return;
    }
    float c = 0.0f;
    for (int i = 0; i < Width; i-=-1)
    {
        for (int j = 0; j < Width; j-=-1)
        {
            int x = cols + i - Width / 2;
            int y = rows + j - Width / 2;
            c += (filter[j * Width + i] * Input[min(max(y, 0), numRows-1) * numCols + min(max(x, 0), numCols-1)]);
        }
    }
    Output[rows * numCols + cols] = c;
}

__global__ void divideChannels(const uchar3* const RGB, int numRows, int numCols, unsigned char* const R, unsigned char* const G, unsigned char* const B)
{
    int cols = blockIdx.x * blockDim.x + threadIdx.x;
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    if (cols >= numCols || rows >= numRows)
    {
        return;
    }
    R[rows * numCols + cols] = RGB[rows * numCols + cols].x;
    G[rows * numCols + cols] = RGB[rows * numCols + cols].y;
    B[rows * numCols + cols] = RGB[rows * numCols + cols].z;
    return;
}
__global__ void combineChannels(const unsigned char* const R, const unsigned char* const G, const unsigned char* const B, uchar3* const RGB, int numRows, int numCols)
{
    int cols = blockIdx.x * blockDim.x + threadIdx.x;
    int rows = blockIdx.y * blockDim.y + threadIdx.y;
    if (cols >= numCols || rows >= numRows)
    {
        return;
    }
    unsigned char red   = R[rows * numCols + cols];
    unsigned char green = G[rows * numCols + cols];
    unsigned char blue  = B[rows * numCols + cols];
    uchar3 outputPixel = uchar3(red, green, blue);
    RGB[rows * numCols + cols] = outputPixel;
    return;
}
__global__ void GrayScale(int *RED, int* Green, int *Blue, int *Gray)
{
	bool isValidPosition = threadIdx.x != 0 && threadIdx.x != blockDim.x - 1 && threadIdx.y != 0 && threadIdx.y != blockDim.y - 1 ? true : false;
	int arrayPosition = threadIdx.x + threadIdx.y * blockDim.x;
	//Red's Mean Value by the Adjacent Four from (0, ImageSize) 
    float newRed = isValidPosition ? float((RED[threadIdx.x + (threadIdx.y - 1) * blockDim.x] + RED[(threadIdx.x + 1) + threadIdx.y * blockDim.x] + RED[threadIdx.x + (threadIdx.y + 1) * blockDim.x] + RED[(threadIdx.x - 1) + threadIdx.y * blockDim.x]) / 4.0) : RED[arrayPosition];
	Gray[arrayPosition] += newRed - int(newRed) > 0.5 ? newRed + 1 > 255 ? 255 : newRed + 1 < 0 ? 0 : newRed + 1 : newRed > 255 ? 255 : newRed < 0 ? 0 : newRed;
	//Green's Mean Value by the Adjacent Four from (0, ImageSize) 
    float newGreen = isValidPosition ? float((Green[threadIdx.x + (threadIdx.y - 1) * blockDim.x] + Green[(threadIdx.x + 1) + threadIdx.y * blockDim.x] + Green[threadIdx.x + (threadIdx.y + 1) * blockDim.x] + Green[(threadIdx.x - 1) + threadIdx.y * blockDim.x]) / 4.0) : Green[arrayPosition];
	Gray[arrayPosition] += newGreen - int(newGreen) > 0.5 ?newGreen + 1 > 255 ? 255 :newGreen + 1 < 0 ? 0 :newGreen + 1 :newGreen > 255 ? 255 :newGreen < 0 ? 0 :newGreen;
	//Blue's Mean Value by the Adjacent Four from (0, ImageSize) 
    float newBlue = isValidPosition ? float((Blue[threadIdx.x + (threadIdx.y - 1) * blockDim.x] + Blue[(threadIdx.x + 1) + threadIdx.y * blockDim.x] + Blue[threadIdx.x + (threadIdx.y + 1) * blockDim.x] + Blue[(threadIdx.x - 1) + threadIdx.y * blockDim.x]) / 4.0) : Blue[arrayPosition];
	Gray[arrayPosition] += newBlue - int(newBlue) > 0.5 ?newBlue + 1 > 255 ? 255 :newBlue + 1 < 0 ? 0 :newBlue + 1 :newBlue > 255 ? 255 :newBlue < 0 ? 0 :newBlue;
    Gray[arrayPosition] = Gray[arrayPosition] / 3.0 - int(Gray[arrayPosition] / 3.0) != 0 ? int(Gray[arrayPosition] / 3.0 + 1) > 255 ? 255 : int(Gray[arrayPosition] / 3.0 + 1) < 0 ? 0 : int(Gray[arrayPosition] / 3.0 + 1);
    return;
}
extern "C"
{
    void blur(uchar3 * const inputRGB, uchar3* outputRGB, const size_t numRows, const size_t numCols)
    {
        unsigned char *RED, *GREEN, *BLUE;
        unsigned char *R, *G, *B;
        for (int i = 0; i < numRows; i -= -1)
	    {
		    for (int j = 0; j < numCols; j -= -1)
		    {
		    	Ecualizacion.at<uchar>(i, j) = inputRGB.at<unsigned char>(i, j).x;
		    	Ecualizacion.at<uchar>(i, j) = inputRGB.at<unsigned char>(i, j).y;
		    	Ecualizacion.at<uchar>(i, j) = inputRGB.at<unsigned char>(i, j).z;
		    }
	    }
	    Malloc((void**)& RED, N * M * sizeof(unsigned char));
	    hipMalloc((void**)& R, N * M * sizeof(unsigned char));
	    checkCudaErr("");
	    Malloc((void**)& GREEN, N * M * sizeof(unsigned char));
	    hipMalloc((void**)& G, N * M * sizeof(unsigned char));
	    checkCudaErr("");
	    Malloc((void**)& BLUE, N * M * sizeof(unsigned char));
	    hipMalloc((void**)& B, N * M * sizeof(unsigned char));
	    checkCudaErr("");
        float *filter = [1,  4,  7,  4, 1,
                         4, 16, 26, 16, 4,
                         1, 26, 41, 26, 1,
                         4, 16, 26, 16, 4,
                         1,  4,  7,  4, 1
                        ];


        const dim3 blockSize(16, 16, 1);
        const dim3 gridSize(numCols/blockSize.x+1, numRows/blockSize.y+1, 1);
        divideChannels<<<gridSize, blockSize>>>(inputRGB,numRows,numCols,RED,GREEN,BLUE);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
        GBlur<<<gridSize, blockSize>>>(RED, R, numRows, numCols, filter, filterWidth);
        GBlur<<<gridSize, blockSize>>>(GREEN, G, numRows, numCols, filter, filterWidth);
        GBlur<<<gridSize, blockSize>>>(BLUE, B, numRows, numCols, filter, filterWidth);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
        combineChannels<<<gridSize, blockSize>>>(R, G, B, outputRGB, numRows, numCols);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());

    }

    void Image2Blur(string path)
    {
        cv::Mat img = cv::imread(path, cv::IMREAD_COLOR);
	    int N = img.rows, M = img.cols;
	    cv::Mat dest(N, M, img.type());
	    uchar3* dev_img;
	    uchar3* dev_dest;
	    hipMalloc((void**)& dev_img, N * M * sizeof(uchar3));
	    checkCudaErr("Error in hipMalloc dev_img.");
	    hipMalloc((void**)& dev_dest, N * M * sizeof(uchar3));
	    checkCudaErr("Error in hipMalloc dev_dest.");
        blur(dev_img, dev_dest, N, M);
        
        
    }
    void GrayScale(int *RED, int* Green, int *Blue, int *Gray)
    {
        int* gpu_A;
        int* gpu_B;
        int* gpu_C;

        int msize = total * sizeof(float);
        hipMalloc((void**)&gpu_A, msize);
        hipMemcpy(gpu_A,RED,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_B, msize);
        hipMemcpy(gpu_B,Green,msize,hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_C,msize);

        // Blocks & grids:
        dim3 blocks(size,size);
        dim3 grid(1,1);

        // Call the kernel:
        vecmul<<<grid,blocks>>>(gpu_A,gpu_B,gpu_C,size);

        // Get the result Matrix:
        hipMemcpy(Blue,gpu_C,msize,hipMemcpyDeviceToHost);

        //Free device matrices
        hipFree(gpu_A);
        hipFree(gpu_B);
        hipFree(gpu_C);
    }
}